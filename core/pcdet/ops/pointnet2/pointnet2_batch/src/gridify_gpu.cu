#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <torch/serialize/tensor.h>
#include <vector>
#include <THC/THC.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include "gridify.h"


#define CHECK_CUDA(x) do { \
  if (!x.type().is_cuda()) { \
    fprintf(stderr, "%s must be CUDA tensor at %s:%d\n", #x, __FILE__, __LINE__); \
    exit(-1); \
  } \
} while (0)
#define CHECK_CONTIGUOUS(x) do { \
  if (!x.is_contiguous()) { \
    fprintf(stderr, "%s must be contiguous tensor at %s:%d\n", #x, __FILE__, __LINE__); \
    exit(-1); \
  } \
} while (0)
#define CHECK_MALLOC(x) do{int ret = x;printf("%s: %d\n",#x,ret);}while(0)

#define CHECK_INPUT(x) CHECK_CUDA(x);CHECK_CONTIGUOUS(x)
#define TOTAL_THREADS 1024

#define ndim 3
#define data_ndim 4

// See: http://codeyarns.com/2011/03/02/how-to-do-error-checking-in-cuda/
inline void
__cuda_check_errors (const char *filename, const int line_number)
{
    hipError_t err = hipDeviceSynchronize ();
    if (err != hipSuccess)
    {
        printf ("CUDA error %i at %s:%i: %s\n",
                err, filename, line_number, hipGetErrorString (err));
        exit (-1);
    }
}

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
    if (err != hipSuccess)
    {
        printf ("CUDA error %i at %s:%i: %s\n",
                err, filename, line_number, hipGetErrorString (err));
        exit (-1);
    }
}

template<typename Dtype>
__global__ void gridify_kernel_build_index(
        int *out_nebidx, Dtype *out_nebidxmsk, Dtype *out_cent,
        Dtype *out_centmsk, int *out_actual_centnum, int *actual_centcount,
        const Dtype *in_data, const int *in_actual_numpoints,
        const int B,
        const int N,
        const int max_o,
        const int P,
        const int kernel_size,
        const int stride,
        const int loc,
        const float *d_coord_shift,
        const float *d_voxel_size,
        const float *d_grid_size,
        const int grid_size_vol,
        const int size,
        int *coor_to_voxelidx,
        int *voxelidx_to_coor,
        int *coor_to_pntidx,
        float *coor_to_locxyzw,
        int *coor_counter
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // index of gpu thread
    int i_batch = index / N;  // index of batch
    if (i_batch >= B) { return; }
    int i_pt = index - N * i_batch;
//     printf("B = %d, i = %d, in_num = %d\n",i_batch,i_pt,in_actual_numpoints[i_batch]);
    if (i_pt < in_actual_numpoints[i_batch]) {
        hiprandState state;
        int coor[ndim];
        const Dtype *p_pt = in_data + index * data_ndim;
        for (int j = 0; j < ndim; j++) {
            int c = floor((p_pt[j] + d_coord_shift[j]) / d_voxel_size[j]);
            if (c < 0 || c >= d_grid_size[j]) {
                return;
            }
            coor[j] = c;
        }
        int coor_indx = coor[2] * (d_grid_size[0] * d_grid_size[1]) + coor[1] * d_grid_size[0] + coor[0];
        int coor_indx_b = i_batch * grid_size_vol + coor_indx;
        int grid_pntidx = atomicAdd(coor_counter + coor_indx_b, 1);
        if (grid_pntidx < P) {
//             printf("coord: {%d,%d,%d}, vid: %d, b_id: %d, p_id: %d,\n",coor[0],coor[1],coor[2],coor_indx_b,i_batch,i_pt);
            coor_to_pntidx[coor_indx_b * P + grid_pntidx] = i_pt;
        } else {
            hiprand_init(index, 0, 0, &state);
            int insrtidx = ceilf(hiprand_uniform(&state) * (grid_pntidx + 1)) - 1;
            if (insrtidx < P) {
                coor_to_pntidx[coor_indx_b * P + insrtidx] = i_pt;
            }
        }
        if (loc == 1) {
            int coor_b_idx = coor_indx_b * data_ndim;
            float weight = p_pt[3];
            atomicAdd(coor_to_locxyzw + coor_b_idx, p_pt[0] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 1, p_pt[1] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 2, p_pt[2] * weight);
            atomicAdd(coor_to_locxyzw + coor_b_idx + 3, weight);
        }


        int voxel_idx = coor_to_voxelidx[coor_indx_b];
        //        printf("grid_size_vol: %d, coor_index: %d, i_batch %d, voxel_idx: %d ; \n", grid_size_vol, coor_index, i_batch, voxel_idx);
        if (voxel_idx == -1) {  // found an empty voxel
            Dtype old_voxel_num = atomicCAS(
                    &coor_to_voxelidx[coor_indx_b],
                    -1, 0
            );
            if (old_voxel_num == -1) {
                // CAS -> old val, if old val is -1
                // if we get -1, this thread is the one who obtain a new voxel
                // so only this thread should do the increase operator below
                int tmp = atomicAdd(actual_centcount + i_batch, 1); // increase the counter, return old counter
                atomicAdd(out_actual_centnum + i_batch, 1); // increase the counter, return old counter
                if (tmp < max_o) {
                    voxelidx_to_coor[i_batch * max_o + tmp] = coor_indx;
                    out_centmsk[i_batch * max_o + tmp] = 1.0; // change center mask to 1 at new occupied voxel
                } else {
                    hiprand_init(index, 0, 0, &state);
                    int insrtidx = ceilf(hiprand_uniform(&state) * (tmp + 1)) - 1;
                    if (insrtidx < max_o) {
                        voxelidx_to_coor[i_batch * max_o + insrtidx] = coor_indx;
                    }
                }
                if (out_actual_centnum[i_batch] > max_o) {
                    out_actual_centnum[i_batch] = max_o;
                }
            }
        }
    }
}

template<typename Dtype>
__global__ void gridify_kernel_query_neighs(
        int *out_nebidx, Dtype *out_nebidxmsk, Dtype *out_cent, Dtype *out_centmsk, int *out_actual_centnum,
        const Dtype *in_data, const int *in_actual_numpoints,
        const int B,
        const int N,
        const int max_o,
        const int P,
        const int kernel_size,
        const int stride,
        const int loc,
        const float *d_coord_shift,
        const float *d_voxel_size,
        const float *d_grid_size,
        const int grid_size_vol,
        const int size,
        int *coor_to_voxelidx,
        int *voxelidx_to_coor,
        int *coor_to_pntidx,
        float *coor_to_locxyzw,
        int *coor_counter,
        int *voxelidx_counter
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // index of gpu thread
    int i_batch = index / max_o;  // index of batch
    if (i_batch >= B) { return; }
    int i_ogrid = index - i_batch * max_o;
    if (i_ogrid < out_actual_centnum[i_batch]) {
        hiprandState state;
        int coor_indx_b, in_data_eleweight, grid_pntidx = 0, insrtidx, idx, idx_b_data;
        float xsum = 0.0, ysum = 0.0, zsum = 0.0, countweightsum = 0, oldweight = 0;
        //        int coor_indx = coor[2] * (d_grid_size[0] * d_grid_size[1])
        //                        + coor[1] * d_grid_size[0] + coor[0];
        int coor = voxelidx_to_coor[index];
        int coor2 = coor / (d_grid_size[0] * d_grid_size[1]);
        int coor1 = (coor - coor2 * (d_grid_size[0] * d_grid_size[1])) / d_grid_size[0];
        int coor0 = coor - coor2 * (d_grid_size[0] * d_grid_size[1]) - coor1 * d_grid_size[0];
        int d_coor, h_coor, w_coor, initID;
        float total_weight = 0;
        int index_data = index * data_ndim;
        int index_P = index * P;
        int coor_indx_b_origin;
        for (int nei_idx = 0; nei_idx < size; nei_idx++) {
            d_coor = nei_idx / (kernel_size * kernel_size) - (kernel_size - 1) / 2 + coor2;
            h_coor = (nei_idx % (kernel_size * kernel_size)) / kernel_size - (kernel_size - 1) / 2 + coor1;
            w_coor = nei_idx % kernel_size - (kernel_size - 1) / 2 + coor0;
            if (d_coor >= 0 && d_coor < d_grid_size[2] && h_coor >= 0 &&
                h_coor < d_grid_size[1] && w_coor >= 0 && w_coor < d_grid_size[0]) {
                coor_indx_b = i_batch * grid_size_vol + d_coor * (d_grid_size[0] * d_grid_size[1])
                              + h_coor * d_grid_size[0] + w_coor;
                if (nei_idx * 2 + 1 == size) { coor_indx_b_origin = coor_indx_b; }
                int amount = min(P, coor_counter[coor_indx_b]);
                for (int j = 0; j < amount; j++) {
                    if (grid_pntidx++ < P) {
                        idx = coor_to_pntidx[coor_indx_b * P + j];
                        if (grid_pntidx == 1) { initID = idx; }
                        idx_b_data = (idx + N * i_batch) * data_ndim;
                        in_data_eleweight = in_data[idx_b_data + 3];
                        out_nebidx[index_P + grid_pntidx - 1] = idx;
                        out_nebidxmsk[index_P + grid_pntidx - 1] = 1.0;
                        total_weight += in_data_eleweight;
                    } else {
                        hiprand_init(index_P * size + grid_pntidx, 0, 0, &state);
                        insrtidx = ceilf(hiprand_uniform(&state) * (grid_pntidx)) - 1;
                        if (insrtidx < P) {
                            oldweight = in_data[(out_nebidx[index_P + insrtidx] + N * i_batch) * data_ndim + 3];
                            idx = coor_to_pntidx[coor_indx_b * P + j];
                            idx_b_data = (idx + N * i_batch) * data_ndim;
                            in_data_eleweight = in_data[idx_b_data + 3];
                            out_nebidx[index_P + insrtidx] = idx;
                            total_weight += (in_data_eleweight - oldweight);
                        }
                    }
                }
            }
        }
        out_cent[index_data + 3] = total_weight;
        if (grid_pntidx < P) {
            for (int j = grid_pntidx; j < P; j++) {
                out_nebidx[index_P + j] = initID;
            }
        }
        if (loc == 1) {
            int coor_indx_b_data = coor_indx_b_origin * data_ndim;
            xsum = coor_to_locxyzw[coor_indx_b_data];
            ysum = coor_to_locxyzw[coor_indx_b_data + 1];
            zsum = coor_to_locxyzw[coor_indx_b_data + 2];
            countweightsum = coor_to_locxyzw[coor_indx_b_data + 3];
            out_cent[index_data] = xsum / countweightsum;
            out_cent[index_data + 1] = ysum / countweightsum;
            out_cent[index_data + 2] = zsum / countweightsum;
        }
    }
}



int grid_query_wrapper_fast(
        at::Tensor nebidx,
        at::Tensor nebidxmsk,
        at::Tensor cent,
        at::Tensor centmsk,
        at::Tensor actual_centnum,
        at::Tensor data,
        at::Tensor actual_numpoints,
        std::vector<float> param_coord_shift,
        std::vector<float> param_grid_size,
        std::vector<float> param_voxel_size,
        std::vector<int> param_kernel_size
) {

    int stride = 1;
    int loc = 1;

    CHECK_INPUT(nebidx);
    CHECK_INPUT(nebidxmsk);
    CHECK_INPUT(cent);
    CHECK_INPUT(centmsk);
    CHECK_INPUT(actual_centnum);
    CHECK_INPUT(data);
    CHECK_INPUT(actual_numpoints);

    const int B = data.size(0);
    const int N = data.size(1);
    const int O = nebidx.size(1);
    const int P = nebidx.size(2);
    printf("B = %d; N = %d; M = %d; K = %d\n",B,N,O,P);

    int *out_nebidx = nebidx.data<int>();
    float *out_nebidxmsk = nebidxmsk.data<float>();
    float *out_cent = cent.data<float>();
    float *out_centmsk = centmsk.data<float>();
    int *out_actual_centnum = actual_centnum.data<int>();

    const float *in_data = data.data<float>();
    const int *in_actual_numpoints = actual_numpoints.data<int>();

    int grid_size_vol = (int) (param_grid_size[0] * param_grid_size[1] * param_grid_size[2]);
    const int size = param_kernel_size[0] * param_kernel_size[0] * param_kernel_size[0];
    printf("grid_vol = %d; kernel_vol = %d\n",grid_size_vol,size);

    float *coord_shift = new float[3];
    float *voxel_size = new float[3];
    float *grid_size = new float[3];
    for (int i = 0; i < 3; ++i) {
        coord_shift[i] = param_coord_shift[i];
        voxel_size[i] = param_voxel_size[i];
        grid_size[i] = param_grid_size[i];
    }
    printf("voxel: {%f,%f,%f}, grid: {%f,%f,%f}, shift: {%f,%f,%f}\n",
        voxel_size[0],voxel_size[1],voxel_size[2],
        grid_size[0],grid_size[1],grid_size[2],
        coord_shift[0],coord_shift[1],coord_shift[2]);

    float *d_coord_shift, *d_voxel_size, *d_grid_size;
    CHECK_MALLOC(hipMalloc(&d_coord_shift, 3 * sizeof(float)));
    CHECK_MALLOC(hipMalloc(&d_voxel_size, 3 * sizeof(float)));
    CHECK_MALLOC(hipMalloc(&d_grid_size, 3 * sizeof(float)));
    hipMemcpy(d_coord_shift, coord_shift, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_voxel_size, voxel_size, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_size, grid_size, 3 * sizeof(float), hipMemcpyHostToDevice);

    float *d_coor_to_locxyzw;
    int *d_coor_to_voxelidx, *d_voxelidx_to_coor, *d_coor_to_pntidx, *d_coor_counter, *d_voxelidx_counter, *actual_centcount;
    // 这里的coor可能需要修改
    CHECK_MALLOC(hipMalloc(&d_coor_to_locxyzw, B * grid_size_vol * 4 * sizeof(float)));
    CHECK_MALLOC(hipMalloc(&d_coor_to_pntidx, B * grid_size_vol * P * sizeof(int)));
    CHECK_MALLOC(hipMalloc(&d_coor_counter, B * grid_size_vol * sizeof(int)));
    CHECK_MALLOC(hipMalloc(&d_coor_to_voxelidx, B * grid_size_vol * sizeof(int)));
    CHECK_MALLOC(hipMalloc(&d_voxelidx_to_coor, B * O * sizeof(int)));
    CHECK_MALLOC(hipMalloc(&d_voxelidx_counter, B * O * sizeof(int)));
    CHECK_MALLOC(hipMalloc(&actual_centcount, B * sizeof(int)));
    hipMemset(d_coor_to_locxyzw, 0, B * grid_size_vol * 4 * sizeof(float));
    hipMemset(d_coor_counter, 0, B * grid_size_vol * sizeof(int));
    hipMemset(d_voxelidx_counter, 0, B * O * sizeof(int));
    hipMemset(d_coor_to_voxelidx, -1, B * grid_size_vol * sizeof(int));
    hipMemset(actual_centcount, 0, B * sizeof(int));
    printf("coor_counter: %zu\n", B * grid_size_vol * sizeof(int));

    const int gridSize = (B * N + TOTAL_THREADS - 1) / TOTAL_THREADS;
    dim3 dimGrid(gridSize);
    dim3 dimBlock(TOTAL_THREADS);
    printf("build idx kernel launch: dimGrid = %d, dimBlock = %d \n", gridSize, TOTAL_THREADS);
    gridify_kernel_build_index<float><<<dimGrid, dimBlock>>>(
            out_nebidx, out_nebidxmsk, out_cent, out_centmsk, out_actual_centnum, actual_centcount, in_data, in_actual_numpoints,
                    B, N, O, P, param_kernel_size[0], stride, loc, d_coord_shift, d_voxel_size, d_grid_size,
                    grid_size_vol, size, d_coor_to_voxelidx, d_voxelidx_to_coor, d_coor_to_pntidx, d_coor_to_locxyzw, d_coor_counter);

    const int o_gridSize = (B * O + TOTAL_THREADS - 1) / TOTAL_THREADS;
    dim3 o_dimGrid(o_gridSize);
    dim3 o_dimBlock(TOTAL_THREADS);
    printf("query neighs kernel launch: dimGrid = %d, dimBlock = %d \n", o_gridSize, TOTAL_THREADS);
    gridify_kernel_query_neighs<float><<<o_dimGrid, o_dimBlock>>>(
            out_nebidx, out_nebidxmsk, out_cent, out_centmsk, out_actual_centnum, in_data, in_actual_numpoints,
                    B, N, O, P, param_kernel_size[0], stride, loc, d_coord_shift, d_voxel_size, d_grid_size,
                    grid_size_vol, size, d_coor_to_voxelidx, d_voxelidx_to_coor, d_coor_to_pntidx, d_coor_to_locxyzw,
                    d_coor_counter, d_voxelidx_counter);
//    MSHADOW_CUDA_POST_KERNEL_CHECK(cuda::gridify_kernel_query_neighs);
//     thrust::device_ptr<int> d_ptr=thrust::device_pointer_cast<int>(d_coor_to_voxelidx);
//     int LENGTH = B * grid_size_vol;
//     thrust::sort(d_ptr,d_ptr+LENGTH, thrust::greater<int>());
    printf("end kernel\n");
//     while(1){}
    delete coord_shift;
    delete voxel_size;
    delete grid_size;
    hipFree(d_coord_shift);
    hipFree(d_voxel_size);
    hipFree(d_grid_size);
    hipFree(d_coor_to_voxelidx);
    hipFree(d_voxelidx_to_coor);
    hipFree(d_coor_to_pntidx);
    hipFree(d_coor_to_locxyzw);
    hipFree(d_coor_counter);
    hipFree(d_voxelidx_counter);
    hipFree(actual_centcount);
    return 1;
}

